#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 32 
#define N 10240

__global__ void sumValues(int* sum, int* numbers, int length) {
  int index = BLOCK_SIZE * blockIdx.x + threadIdx.x;
  atomicAdd()
}


int main() {
  int* cpu_arr = (int*)malloc(N * sizeof(int));
  if(!cpu_arr) {
    perror("malloc");
    exit(1);
  }  

  for(int i = 0; i < N; i++) {
    cpu_arr[i] = i;
  }

  int* gpu_arr;

  if(hipMalloc(&gpu_arr, sizeof(int) * N) != hipSuccess) {
    fprintf(stderr, "Failed to allocate array on GPU\n");
    exit(2);
  }

  if(hipMemcpy(gpu_arr, cpu_arr, sizeof(int) * N, hipMemcpyHostToDevice) != hipSuccess) {
    fprintf(stderr, "Failed to copy array to the GPU\n");
  }

  int *sum;
  if(hipMalloc(&sum, sizeof(int)) != hipSuccess) {
    fprintf(stderr, "Failed to allocate array on GPU\n");
    exit(2);
  }

  sumValues<<<N/BLOCK_SIZE, BLOCK_SIZE>>>(sum, gpu_arr, N);
  hipDeviceSynchronize();

  if(hipMemcpy(cpu_arr, gpu_difference, sizeof(int) * N, hipMemcpyDeviceToHost) != hipSuccess) {
    fprintf(stderr, "Failed to copy array to the CPU\n");
  }
  
  for(int i = 0; i < N; i++) {
    printf("%d\n", cpu_arr[i]);
  }

  free(cpu_arr);
  hipFree(gpu_arr);
  hipFree(gpu_difference);

  return 0;

}

#include "hip/hip_runtime.h"
#include <math.h>
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>

#include <SDL.h>

#include "gui.h"

// Time step size
#define DT 0.075

// Gravitational constant
#define G 100

#define BLOCK_SIZE 32

// This struct holds data for a single star
typedef struct star {
  float x_position;
  float y_position;
  float x_velocity;
  float y_velocity;
  float mass;
} star_t;

// Generate a random float in a given range
float drand(float min, float max) {
  return ((float)rand() / RAND_MAX) * (max - min) + min;
}

// Compute the radius of a star based on its mass
__device__ __host__ float star_radius(float mass) {
  return sqrt(mass);
}

__global__ void updatePosition(star_t* stars, int num_stars) {
  int index = BLOCK_SIZE * blockIdx.x + threadIdx.x;
  if(index < num_stars) {
    stars[index].x_position += stars[index].x_velocity * DT;
    stars[index].y_position += stars[index].y_velocity * DT;
    // Loop over all other stars to compute forces
    for(int j=0; j<num_stars; j++) {
      // Don't compute the force of a star on itself
      if(index == j) continue;

      // Compute the distance between the two stars in each dimension
      float x_diff = stars[index].x_position - stars[j].x_position;
      float y_diff = stars[index].y_position - stars[j].y_position;

      // Compute the magnitude of the distance vector
      float dist = sqrt(x_diff * x_diff + y_diff * y_diff);

      // Normalize the distance vector components
      x_diff /= dist;
      y_diff /= dist;

      // Keep a minimum distance, otherwise we get 
      float combined_radius = star_radius(stars[index].mass) + star_radius(stars[j].mass);
      if(dist < combined_radius) {
        dist = combined_radius;
      }

      // Compute the x and y accelerations
      float x_acceleration = -x_diff * G * stars[j].mass / (dist * dist);
      float y_acceleration = -y_diff * G * stars[j].mass / (dist * dist);

      // Update the star velocity
      stars[index].x_velocity += x_acceleration * DT;
      stars[index].y_velocity += y_acceleration * DT;

      // Handle edge collisiosn
      if(stars[index].x_position < 0 && stars[index].x_velocity < 0)
        stars[index].x_velocity *= -0.5;
      if(stars[index].x_position >= SCREEN_WIDTH && stars[index].x_velocity
          > 0) stars[index].x_velocity *= -0.5;
      if(stars[index].y_position < 0 && stars[index].y_velocity < 0) stars[index].y_velocity *= -0.5;
      if(stars[index].y_position >= SCREEN_HEIGHT && stars[index].y_velocity
          > 0) stars[index].y_velocity *= -0.5;
    }
  }
}



int main(int argc, char** argv) {
  // Initialize the graphical interface
  gui_init();

  // Run as long as this is true
  bool running = true;

  // Is the mouse currently clicked?
  bool clicked = false;

  // This will hold our array of stars
  star_t* stars = NULL;
  int num_stars = 0;
  star_t* gpu_stars = NULL;
  int prev_num_stars = -1;

  // Start main loop
  while(running) {
    // Check for events
    SDL_Event event;
    while(SDL_PollEvent(&event) == 1) {
      // If the event is a quit event, then leave the loop
      if(event.type == SDL_QUIT) running = false;
    }

    // Get the current mouse state
    int mouse_x, mouse_y;
    uint32_t mouse_state = SDL_GetMouseState(&mouse_x, &mouse_y);

    // Is the mouse pressed?
    if(mouse_state & SDL_BUTTON(SDL_BUTTON_LEFT)) {
      // Is this the beginning of a mouse click?
      if(!clicked) {
        clicked = true;
        stars = (star_t*)realloc(stars, (num_stars + 1) * sizeof(star_t));
        stars[num_stars].x_position = mouse_x;
        stars[num_stars].y_position = mouse_y;
        stars[num_stars].x_velocity = 0;
        stars[num_stars].y_velocity = 0;
        // Generate a random mass skewed toward small sizes
        stars[num_stars].mass = drand(0, 1) * drand(0, 1) * 50;
        num_stars++;
      }
    } else {
      // The mouse click is finished
      clicked = false;
    }
    if(prev_num_stars != num_stars) {
      if(prev_num_stars != -1) {
        hipFree(gpu_stars);
      }
      if(hipMalloc(&gpu_stars, sizeof(star_t) * num_stars) != hipSuccess) {  
        fprintf(stderr, "Failed to allocate X array on GPU\n");
        exit(2);
      }
      prev_num_stars = num_stars;
    }

    // Draw stars
    for(int i=0; i<num_stars; i++) {
      color_t color = {255, 255, 255, 255};
      gui_draw_circle(stars[i].x_position, stars[i].y_position, star_radius(stars[i].mass), color);
    }

    // Calculate the number of blocks we need
    size_t blocks = (num_stars + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Copy from CPU to GPU
    if(hipMemcpy(gpu_stars, stars, sizeof(star_t) * num_stars, hipMemcpyHostToDevice) != hipSuccess) {
      fprintf(stderr, "Failed to copy stars to the GPU\n");
    }
    // Update the position in GPU
    updatePosition<<<blocks, BLOCK_SIZE>>>(gpu_stars, num_stars);
    // Wait untill all threads join
    hipDeviceSynchronize();

    // Copy from GPU to CPU
    if(hipMemcpy(stars,gpu_stars, sizeof(star_t) * num_stars,
          hipMemcpyDeviceToHost) != hipSuccess) {
      fprintf(stderr, "Failed to copy gpu_stars from the GPU\n");
    }

    // Display the rendered image
    gui_update_display();

    // Fade out the rendered image to leave movement trails
    gui_fade(0.85);
  }

  // Free the stars array
  free(stars);

  // Clean up the graphical interface
  gui_shutdown();

  return 0;
}

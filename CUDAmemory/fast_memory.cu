
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 32 
#define N 10240

__global__ void doubleValues(int*difference, int* numbers, int length) {
  __shared__ int local_values[N];
  int index = BLOCK_SIZE * blockIdx.x + threadIdx.x;
  local_values[index] = numbers[index];
  __syncthreads();
  if(index != length - 1) {
    difference[index] = local_values[index + 1] - local_values[index]; 
  }
}


int main() {
  int* cpu_arr = (int*)malloc(N * sizeof(int));
  if(!cpu_arr) {
    perror("malloc");
    exit(1);
  }  

  for(int i = 0; i < N; i++) {
    cpu_arr[i] = i * i;
  }

  int* gpu_arr;

  if(hipMalloc(&gpu_arr, sizeof(int) * N) != hipSuccess) {
    fprintf(stderr, "Failed to allocate array on GPU\n");
    exit(2);
  }

  if(hipMemcpy(gpu_arr, cpu_arr, sizeof(int) * N, hipMemcpyHostToDevice) != hipSuccess) {
    fprintf(stderr, "Failed to copy array to the GPU\n");
  }

  int* gpu_difference;
  if(hipMalloc(&gpu_difference, sizeof(int) * N) != hipSuccess) {
    fprintf(stderr, "Failed to allocate array on GPU\n");
    exit(2);
  }

  doubleValues<<<N/BLOCK_SIZE, BLOCK_SIZE>>>(gpu_difference, gpu_arr, N);
  hipDeviceSynchronize();

  if(hipMemcpy(cpu_arr, gpu_difference, sizeof(int) * N, hipMemcpyDeviceToHost) != hipSuccess) {
    fprintf(stderr, "Failed to copy array to the CPU\n");
  }
  
  for(int i = 0; i < N; i++) {
    printf("%d\n", cpu_arr[i]);
  }

  free(cpu_arr);
  hipFree(gpu_arr);
  hipFree(gpu_difference);

  return 0;

}
